#include "hip/hip_runtime.h"
/*  -*- mode: c++ -*-  */
#include "gg.h"
#include "ggcuda.h"

void kernel_sizing(CSRGraph &, dim3 &, dim3 &);
#define TB_SIZE 256
const char *GGC_OPTIONS = "coop_conv=False $ outline_iterate_gb=False $ backoff_blocking_factor=4 $ parcomb=False $ np_schedulers=set(['fg', 'tb', 'wp']) $ cc_disable=set([]) $ hacks=set([]) $ np_factor=1 $ instrument=set([]) $ unroll=[] $ instrument_mode=None $ read_props=None $ outline_iterate=True $ ignore_nested_errors=False $ np=False $ write_props=None $ quiet_cgen=True $ retry_backoff=True $ cuda.graph_type=basic $ cuda.use_worklist_slots=True $ cuda.worklist_type=basic";
AppendOnlyList el;
#include "mst.h"
#define INF UINT_MAX
const int DEBUG = 0;
static const int __tb_union_components = TB_SIZE;
__global__ void init_wl(CSRGraph graph, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type node_end;
  // FP: "1 -> 2;
  node_end = (graph).nnodes;
  for (index_type node = 0 + tid; node < node_end; node += nthreads)
  {
    (out_wl).push(node);
  }
  // FP: "4 -> 5;
}
__global__ void find_comp_min_elem(CSRGraph graph, struct comp_data comp, LockArrayTicket complocks, ComponentSpace cs, int level, AppendOnlyList bosses, Worklist2 in_wl, Worklist2 out_wl)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlnode_end;
  // FP: "1 -> 2;

  // FP: "2 -> 3;
  wlnode_end = *((volatile index_type *) (in_wl).dindex);
  for (index_type wlnode = 0 + tid; wlnode < wlnode_end; wlnode += nthreads)
  {
    int node;
    bool pop;
    index_type edge_end;
    pop = (in_wl).pop_id(wlnode, node);
    unsigned minwt = INF;
    unsigned minedge = INF;
    int degree = graph.getOutDegree(node);
    int mindstcomp  = 0;
    int srccomp = cs.find(node);
    bool isBoss = srccomp == node;
    edge_end = (graph).getFirstEdge((node) + 1);
    for (index_type edge = (graph).getFirstEdge(node) + 0; edge < edge_end; edge += 1)
    {
      int edgewt = graph.getAbsWeight(edge);
      if (edgewt < minwt)
      {
        int dstcomp = cs.find(graph.getAbsDestination(edge));
        if (dstcomp != srccomp)
        {
          minwt = edgewt;
          minedge = edge;
        }
      }
    }
    if (isBoss && degree)
    {
      bosses.push(node);
    }
    if (minwt != INF)
    {
      (out_wl).push(node);
      {
        #if __CUDACC_VER_MAJOR__ >= 7
        volatile bool done_ = false;
        #else
        bool done_ = false;
        #endif
        int _ticket = (complocks).reserve(srccomp);
        while (!done_)
        {
          if (complocks.acquire_or_fail(srccomp, _ticket))
          {
            if (comp.minwt[srccomp] == 0 || (comp.lvl[srccomp] < level) || (comp.minwt[srccomp] > minwt))
            {
              comp.minwt[srccomp] = minwt;
              comp.lvl[srccomp] = level;
              comp.minedge[srccomp] = minedge;
            }
            complocks.release(srccomp);
            done_ = true;
          }
        }
      }
    }
    else
    {
      if (isBoss && degree)
      {
        (out_wl).push(node);
      }
    }
  }
  // FP: "30 -> 31;
}
__global__ void union_components(CSRGraph graph, ComponentSpace cs, struct comp_data compdata, int level, AppendOnlyList el, AppendOnlyList ew, AppendOnlyList b_in, AppendOnlyList b_out, Worklist2 in_wl, Worklist2 out_wl, GlobalBarrier gb, Any ret_val)
{
  unsigned tid = TID_1D;
  unsigned nthreads = TOTAL_THREADS_1D;

  const unsigned __kernel_tb_size = TB_SIZE;
  if (tid == 0)
    in_wl.reset_next_slot();

  index_type wlnode_end;
  index_type wlnode_rup;
  // FP: "1 -> 2;
  wlnode_end = *((volatile index_type *) (b_in).dindex);
  wlnode_rup = ((0) + roundup(((*((volatile index_type *) (b_in).dindex)) - (0)), (nthreads)));
  for (index_type wlnode = 0 + tid; wlnode < wlnode_rup; wlnode += nthreads)
  {
    int node;
    bool pop;
    pop = (b_in).pop_id(wlnode, node);
    int r = 0;
    int dstcomp = -1;
    int srccomp = -1;
    if (pop && compdata.lvl[node] == level)
    {
      srccomp = cs.find(node);
      dstcomp = cs.find(graph.getAbsDestination(compdata.minedge[node]));
    }
    gb.Sync();
    if (srccomp != dstcomp)
    {
      if (!cs.unify(srccomp, dstcomp))
      {
        b_out.push(node);
        r = 1;
      }
      else
      {
        el.push(compdata.minedge[node]);
        ew.push(compdata.minwt[node]);
      }
    }
    gb.Sync();
    if (r)
    {
      ret_val.return_(true);
      continue;
    }
  }
}
void gg_main(CSRGraph& hg, CSRGraph& gg)
{
  dim3 blocks, threads;
  kernel_sizing(gg, blocks, threads);
  static GlobalBarrierLifetime union_components_barrier;
  static bool union_components_barrier_inited;
  struct comp_data comp;
  PipeContextT<Worklist2> pipe;
  // FP: "1 -> 2;
  ComponentSpace cs (hg.nnodes);
  // FP: "2 -> 3;
  el = AppendOnlyList(hg.nedges);
  // FP: "3 -> 4;
  AppendOnlyList ew (hg.nedges);
  // FP: "4 -> 5;
  AppendOnlyList bosses[2] = {AppendOnlyList(hg.nnodes), AppendOnlyList(hg.nnodes)};
  int cur_boss = 0;
  // FP: "5 -> 6;
  static const size_t union_components_residency = maximum_residency(union_components, __tb_union_components, 0);
  static const size_t union_components_blocks = GG_MIN(blocks.x, ggc_get_nSM() * union_components_residency);
  if(!union_components_barrier_inited) { union_components_barrier.Setup(union_components_blocks); union_components_barrier_inited = true;};
  // FP: "6 -> 7;
  // FP: "7 -> 8;
  comp.weight.alloc(hg.nnodes);
  comp.edge.alloc(hg.nnodes);
  comp.node.alloc(hg.nnodes);
  comp.level.alloc(hg.nnodes);
  comp.dstcomp.alloc(hg.nnodes);
  comp.lvl = comp.level.zero_gpu();
  comp.minwt = comp.weight.zero_gpu();
  comp.minedge = comp.edge.gpu_wr_ptr();
  comp.minnode = comp.node.gpu_wr_ptr();
  comp.mindstcomp = comp.dstcomp.gpu_wr_ptr();
  // FP: "8 -> 9;
  LockArrayTicket complocks (hg.nnodes);
  // FP: "9 -> 10;
  int level = 1;
  int mw = 0;
  int last_mw = 0;
  // FP: "10 -> 11;
  pipe = PipeContextT<Worklist2>(hg.nnodes);
  {
    {
      pipe.out_wl().will_write();
      init_wl <<<blocks, threads>>>(gg, pipe.in_wl(), pipe.out_wl());
      pipe.in_wl().swap_slots();
      pipe.advance2();
      // FP: "12 -> 13;
      while (pipe.in_wl().nitems())
      {
        bool loopc = false;
        last_mw = mw;
        pipe.out_wl().will_write();
        find_comp_min_elem <<<blocks, threads>>>(gg, comp, complocks, cs, level, bosses[cur_boss], pipe.in_wl(), pipe.out_wl());
        pipe.in_wl().swap_slots();
        pipe.advance2();
        do
        {
          Shared<int> retval = Shared<int>(1);
          Any _rv;
          *(retval.cpu_wr_ptr()) = 0;
          _rv.rv = retval.gpu_wr_ptr();
          pipe.out_wl().will_write();
          union_components <<<union_components_blocks, __tb_union_components>>>(gg, cs, comp, level, el, ew, bosses[cur_boss], bosses[cur_boss ^ 1], pipe.in_wl(), pipe.out_wl(), union_components_barrier, _rv);
          loopc = *(retval.cpu_rd_ptr()) > 0;
          cur_boss ^= 1;
          bosses[cur_boss].reset();
        }
        while (loopc);
        mw = el.nitems();
        level++;
        if (last_mw == mw)
        {
          break;
        }
      }
      // FP: "23 -> 24;
    }
  }
  pipe.free();
  // FP: "11 -> 12;
  unsigned long int rweight = 0;
  size_t nmstedges ;
  // FP: "24 -> 25;
  nmstedges = ew.nitems();
  mgpu::Reduce(ew.list.gpu_rd_ptr(), nmstedges, (long unsigned int)0, mgpu::plus<long unsigned int>(), (long unsigned int*)0, &rweight, *mgc);
  // FP: "25 -> 26;
  printf("final mstwt: %llu\n", rweight);
  printf("total edges: %llu, total components: %llu\n", nmstedges, cs.numberOfComponentsHost());
  // FP: "26 -> 27;
}